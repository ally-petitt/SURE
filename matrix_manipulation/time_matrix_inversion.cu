#include <cstdio>
#include <cstdlib>
#include <vector>
#include <time.h>
#include <iostream>
#include <fstream>

#include <hipblas.h>
#include <hip/hip_runtime.h>


using data_type = double;


std::vector<data_type> generateRandomMatrix(int size) {
    // Calculate total number of elements in the matrix
    int totalElements = size * size;

    // Create a vector to store the flattened matrix
    std::vector<data_type> matrix(totalElements);

    // Seed for random number generator
    //std::srand(static_cast<unsigned int>(std::time(nullptr)));

    // Initialize matrix with random values
    for (int i = 0; i < totalElements; ++i) {
        matrix[i] = static_cast<data_type>(std::rand() % 100); // Random integers (0-99)
    }

    return matrix;
}


int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    
	// create our output file
    FILE *output_file = fopen("matrix_inversion_short.csv", "w");	
    if (!output_file) {
        std::cerr << "Failed to open the file!" << std::endl;
        return 1;
    }
    
    fprintf(output_file, "Matrix Size, Execution Time (seconds)\n");


    for (int i = 1; i <= 85; i++) {
	    int n = 50 * i;
	    printf("Matrix size: %d: ", n);
	    

	    const std::vector<data_type> A = generateRandomMatrix(n); // initialize source matrix
	    const std::vector<data_type> C = generateRandomMatrix(n); // initialize destination matrix
        const std::vector<data_type> Info = generateRandomMatrix((int) n); // vector containing info on our matrix inversions


	    data_type *d_A = nullptr;
	    data_type *d_C = nullptr;
        data_type *d_Info = nullptr;

	    /* step 1: create cublas handle, bind a stream */
	    (hipblasCreate(&cublasH));

	    (hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	    (hipblasSetStream(cublasH, stream));

	    /* step 2: copy data to device */
	    (hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
	    (hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(data_type) * C.size()));
        (hipMalloc(reinterpret_cast<void **>(&d_Info), sizeof(data_type) * Info.size()));

	    (hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
				       stream));
        (hipMemcpyAsync(d_Info, Info.data(), sizeof(data_type) * Info.size(), hipMemcpyHostToDevice,
				       stream));

	    /* step 3: compute */
	    clock_t start = clock();
	    (cublasDmatinvBatched(cublasH, 
                            n, 
                            (const double *const *) d_A, 
                            n, 
                            (double *const *)d_C, 
                            n, 
                            (int *)d_Info, 
                            n));

	    /* step 4: copy data to host */
	    (hipMemcpyAsync((void *)C.data(), d_C, sizeof(data_type) * C.size(), hipMemcpyDeviceToHost,
				       stream));

        hipStreamSynchronize(stream);
	    clock_t end = clock();
        long double num_seconds = (long double)(end - start) / CLOCKS_PER_SEC;

	     printf(" %Lf seconds\n",  num_seconds);    
	    

	    /* free resources */
	    (hipFree(d_A));
	    (hipFree(d_Info));
        hipFree(d_C);

        

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            // Handle error appropriately
        }
	            
	    fprintf(output_file, "%d,%Lf\n", n, num_seconds);

    }

    // clean up used resources
    (hipblasDestroy(cublasH));

    (hipStreamDestroy(stream));

    (hipDeviceReset());
    fclose(output_file);

    return EXIT_SUCCESS;
}
